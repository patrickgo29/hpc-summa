#include "hip/hip_runtime.h"
#include "cudamultiply.h"

__global__ void kernelFunc(int m, int n, int k, float* ad, float* bd, float* cd) {
    double v = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int ind;
    for (ind = 0; ind < k; ++ind)
    {
       v += ad[row+ind*m]*bd[ind+col*k];
    }

   cd[row+m*col] += Ctemp + cd[row+m*col];
   __syncthreads();
}

void matrix_multiply_cuda(int m, int n, int k,
	      const double* A, int lda, const double* B, int ldb,
	      double* C, int ldc) {
		  
	assert (A || m <= 0 || k <= 0); assert (lda >= m);
	assert (B || k <= 0 || n <= 0); assert (ldb >= k);
	assert (C || m <= 0 || n <= 0); assert (ldc >= m);	  
	
    float* ad;
    float* bd;
    float* cd;
    
    hipMalloc((void**)&ad, m * k * sizeof(float));
    hipMalloc((void**)&bd, k * n * sizeof(float));
    hipMalloc((void**)&cd, m * n * sizeof(float));
    
    hipMemcpy(ad, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bd, B, k * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cd, C, m * n * sizeof(float), hipMemcpyHostToDevice);

	// What dimension?
    dim3 block(?, ?);           
    dim3 grid(?, ?);
    
    kernelFunc<<<grid, block>>>(m,n,k,ad, bd, cd);

    hipMemcpy(c, cd, m * n * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
}